#include "hip/hip_runtime.h"
#include "reducescatter.h"

using namespace hipcub;

__global__ void kernel(value *v1d, value *v2d, value *sep, uint n1, uint n2) {

	/*
	assert(n1 >= THREADS * ITEMS);
	assert(n1 % (THREADS * ITEMS) == 0);
	assert(n2 >= THREADS * ITEMS);
	assert(n2 % (THREADS * ITEMS) == 0);
	*/

	register uint tx = threadIdx.x, bx = blockIdx.x;
	register uint i, m1 = n1 / THREADS, m2 = n2 / THREADS;
	typedef BlockReduce<value, THREADS, ALGORITHM> BlockReduceT;
	__shared__ typename BlockReduceT::TempStorage shared;
	__shared__ value sepratio;
	register value reduction = 0; // Identity element for reduction
	register value data[1];

	for (i = 0; i < m1; i++) {
		LOADDIRECTBLOCKED(tx, v1d + bx * n1 + i * THREADS, data);
		reduction += BlockReduceT(shared).Sum(data);
		__syncthreads();
	}

	if ((m1 = n1 % THREADS)) {
		LoadDirectBlocked(tx, v1d + bx * n1 + i * THREADS, data, m1);
		reduction += BlockReduceT(shared).Sum(data[0], m1);
		__syncthreads();
	}

	if (!tx) {
		sepratio = max((value)0, reduction / sep[bx]); // Old separator value
		sep[bx] = reduction; // Update separator
	}
	__syncthreads();

	for (i = 0; i < m2; i++) {
		LOADDIRECTBLOCKED(tx, v2d + bx * n2 + i * THREADS, data);
		data[0] *= sepratio;
		STOREDIRECTBLOCKED(tx, v2d + bx * n2 + i * THREADS, data);
	}

	if ((m2 = n2 % THREADS)) {
		LoadDirectBlocked(tx, v2d + bx * n2 + i * THREADS, data, m2);
		if (tx < m2) data[0] *= sepratio;
		StoreDirectBlocked(tx, v2d + bx * n2 + i * THREADS, data, m2);
	}
}

void updatepotential(func *f1, func *f2, func *sep, const dim *domains, value *f2sum, value *sepsum, float *transfer, float *preprocess, float *reducescatter) {

	#ifdef MEASURETIME
	struct timeval t1, t2;
	gettimeofday(&t1, NULL);
	#endif

	chunk c1, c2;
	sharedmasks(f1, &c1, f2, &c2);
	shared2most(f1, c1);
	reordershared(f1, f2);

	#ifdef MEASURETIME
	gettimeofday(&t2, NULL);
	(*preprocess) += (float)(t2.tv_usec - t1.tv_usec) / 1e3 + (float)(t2.tv_sec - t1.tv_sec) * 1e3;
	#endif

	value *v1d, *v2d, *sd;
	hipMalloc(&v1d, sizeof(value) * f1->n);
        hipMalloc(&v2d, sizeof(value) * f2->n);
        hipMalloc(&sd, sizeof(value) * sep->n);
	dim n1 = domainproduct(f1->vars, f1->m - f1->s, domains);
	dim n2 = domainproduct(f2->vars, f2->m - f1->s, domains);

	#ifdef STREAMS
	dim ns = CEIL(sep->n, MAXBLOCKS);
	//printf("Each of the %u blocks has to reduce %u rows and product %u rows\n", sep->n, n1, n2);
	hipStream_t *streams = (hipStream_t *)malloc(sizeof(hipStream_t) * ns);
	dim *blocks = (dim *)malloc(sizeof(dim) * ns);
	for (dim i = 0; i < ns - 1; i++) blocks[i] = MAXBLOCKS;
	blocks[ns - 1] = (sep->n % MAXBLOCKS) ? (sep->n % MAXBLOCKS) : MAXBLOCKS;
	//printbuf(blocks, ns, "Blocks");
	#endif

	#ifdef STREAMS
	for (dim i = 0; i < ns; i++) {
		hipStreamCreate(streams + i);
		hipMemcpyAsync(v1d + i * MAXBLOCKS * n1, f1->v + i * MAXBLOCKS * n1, sizeof(value) * n1 * blocks[i], hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(v2d + i * MAXBLOCKS * n2, f2->v + i * MAXBLOCKS * n2, sizeof(value) * n2 * blocks[i], hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(sd + i * MAXBLOCKS, sep->v + i * MAXBLOCKS, sizeof(value) * blocks[i], hipMemcpyHostToDevice, streams[i]);
	}
	#else

	#ifdef MEASURETIME
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	#endif

	hipMemcpy(v1d, f1->v, sizeof(value) * n1 * sep->n, hipMemcpyHostToDevice);
	hipMemcpy(v2d, f2->v, sizeof(value) * n2 * sep->n, hipMemcpyHostToDevice);
	hipMemcpy(sd, sep->v, sizeof(value) * sep->n, hipMemcpyHostToDevice);

	#ifdef MEASURETIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	*transfer += elapsed;
	#endif
	#endif

	#ifdef STREAMS
	for (dim i = 0; i < ns; i++)
		kernel<<<blocks[i], THREADS, 0, streams[i]>>>(v1d + MAXBLOCKS * i * n1, v2d + MAXBLOCKS * i * n2, sd + MAXBLOCKS * i, n1, n2);
	#else

	#ifdef MEASURETIME
	hipEventRecord(start);
	#endif

	kernel<<<sep->n, THREADS>>>(v1d, v2d, sd, n1, n2);

	#ifdef MEASURETIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	*reducescatter += elapsed;
	#endif

	#ifdef TABLESUM
	value *f2sumd;
	hipMalloc(&f2sumd, sizeof(value));
	void *ts = NULL;
	size_t tsn = 0;
	hipcub::DeviceReduce::Sum(ts, tsn, v2d, f2sumd, f2->n);
	hipMalloc(&ts, tsn);
	hipcub::DeviceReduce::Sum(ts, tsn, v2d, f2sumd, f2->n);
	hipFree(ts);
	hipMemcpy(f2sum, f2sumd, sizeof(value), hipMemcpyDeviceToHost);
	hipFree(f2sumd);

	value *sepsumd;
	hipMalloc(&sepsumd, sizeof(value));
	ts = NULL;
	tsn = 0;
	hipcub::DeviceReduce::Sum(ts, tsn, sd, sepsumd, sep->n);
	hipMalloc(&ts, tsn);
	hipcub::DeviceReduce::Sum(ts, tsn, sd, sepsumd, sep->n);
	hipFree(ts);
	hipMemcpy(sepsum, sepsumd, sizeof(value), hipMemcpyDeviceToHost);
	hipFree(sepsumd);
	#endif
	#endif

	gettimeofday(&t1, NULL);

	#ifdef STREAMS
	for (dim i = 0; i < ns; i++) {
		hipMemcpyAsync(f2->v + i * MAXBLOCKS * n2, v2d + i * MAXBLOCKS * n2, sizeof(value) * n2 * blocks[i], hipMemcpyDeviceToHost, streams[i]);
		hipMemcpyAsync(sep->v + i * MAXBLOCKS, sd + i * MAXBLOCKS, sizeof(value) * blocks[i], hipMemcpyDeviceToHost, streams[i]);
	}
	#else

	#ifdef MEASURETIME
	hipEventRecord(start);
	#endif

	hipMemcpy(f2->v, v2d, sizeof(value) * n2 * sep->n, hipMemcpyDeviceToHost);
	hipMemcpy(sep->v, sd, sizeof(value) * sep->n, hipMemcpyDeviceToHost);

	#ifdef MEASURETIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	*transfer += elapsed;
	#endif
	#endif

	CubDebugExit(hipPeekAtLastError());
	CubDebugExit(hipDeviceSynchronize());

	hipFree(v1d);
	hipFree(v2d);
	hipFree(sd);
	#ifdef STREAMS
	free(streams);
	free(blocks);
	#endif
}
